
#include <hip/hip_runtime.h>
# include <iostream>
# define N 10000 * 33
# define blocksPerGrid 32

# define threadsPerBlock 256

__global__ void matMult(float *a, float *b, float *c)
{
    __shared__ float cache[threadsPerBlock];
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;
    while(id < N)
    {
        temp += a[id] * b[id];
        id += blockDim.x * gridDim.x;
    }
    
    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x / 2;

    while(i != 0)
    {
        if(cacheIndex < i)
        {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();

        i = i / 2;
    }

    if(cacheIndex == 0)
    {
        c[blockIdx.x] = cache[0];
    }
}

int main(void)
{
    float *A, *B, *partial_c;

    float *device_a, *device_b, *device_c;

    float C = 0;

    A = (float*)malloc(N * sizeof(float));
    B = (float*)malloc(N * sizeof(float));
    partial_c = (float*)malloc(blocksPerGrid * sizeof(float));
    hipMalloc((void**)&device_a, N * sizeof(float));
    hipMalloc((void**)&device_b, N * sizeof(float));
    hipMalloc((void**)&device_c, blocksPerGrid * sizeof(float));

    for(int i = 0; i < N; i++)
    {
        A[i] = i;
        B[i] = i;
    }

    hipMemcpy(device_a, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b, B, N * sizeof(float), hipMemcpyHostToDevice);

    matMult<<<blocksPerGrid, threadsPerBlock>>>(device_a, device_b, device_c);

    hipMemcpy(partial_c, device_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < blocksPerGrid; i++)
    {
        C += partial_c[i];
    }

    printf("result: %f\n", C);
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
    free(A);
    free(B);
    free(partial_c);
    return 0;
}