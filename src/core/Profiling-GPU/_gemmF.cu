
#include <hip/hip_runtime.h>
# include <iostream>
# define count 1024

__global__ void gemmGPU(int *a, int *b, int *c, int M, int N, int K)
{
    for(int row = threadIdx.x + blockIdx.x * blockDim.x; row < M; row += blockDim.x * gridDim.x)
    {
        for(int col = threadIdx.y + blockIdx.y * blockDim.y; col < N; col += blockDim.y * gridDim.y)
        {
            for(int k = 0; k < K; k++)
            {
                c[ row * N + col ] += a[row * K + k] * b[k * N + col];
            }
            
        }
    }
}
void printMatrix(int *a, int M, int N)
{
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++)
        {
            printf(" %d ", a[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void gemmCPU(int *a, int *b, int *c, int M, int N, int K)
{
    for(int i = 0; i < M; i++)
    {
        for(int j = 0; j < N; j++)
        {
            for(int k = 0; k < K; k++)
            {
                c[i * N + j] += a[i * K + k] * b[k * N + j];
            }
        }
    }
}

int main(void)
{
    int *a, *b, *c;

    size_t size = count * count * sizeof(int);

    a = (int*) malloc(size);
    b = (int*) malloc(size);
    c = (int*) malloc(size);

    for (int i = 0;  i < count * count; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    gemmCPU(a, b, c, count, count, count);

    int *d = (int*) malloc(size);

    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((count+15)/16, (count+15)/16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    gemmGPU<<<gridDim, blockDim>>>(d_a, d_b, d_c, count, count, count);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel elapsed time: %f \n", milliseconds);

    hipMemcpy(d, d_c, size, hipMemcpyDeviceToHost);

    if(memcmp(c, d, size) == 0)
    {
        printf("Winner winner chicken dinner!\n");
    }
    else
    {
        printf("Fuck you loser\n");
    }

    free(a);
    free(b);
    free(c);
    free(d);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;

}