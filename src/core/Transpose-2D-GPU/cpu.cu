
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
# define N 1000 
#include <iomanip>

__global__ void d_transpose(int *d_a, int *d_c, int rows, int columns)
{
    // here even if we flip it, shit doesnt matter cus at the end of the day all we need is
    // every single combination of row id and column id for atleast one thread to do thecorresponding calculation for an accurate transpose.
    int columnId = threadIdx.x + blockDim.x * blockIdx.x;
    int rowId = threadIdx.y + blockDim.y * blockIdx.y;

    while(rowId < rows && columnId < columns)
    {
        d_c[rowId * columns + columnId] = d_a[columnId * rows + rowId];

        columnId += gridDim.x * blockDim.x;
        rowId += gridDim.y * blockDim.y;
    }
}

void print_matrix(const int *matrix, int rows, int cols, const std::string& name) {
    std::cout << name << ":\n";
    for (int row = 0; row < rows; row++) {
        for (int col = 0; col < cols; col++) {
            // i think this is pretty simple too tbh, i could write this myself next time for repeittion tmrw
            // all it is doing is printing each element in row major order sequentially with spaces and line breaks
            std::cout << std::setw(4) << matrix[row * cols + col] << " ";
        }
        std::cout << "\n";
    }
    std::cout << std::endl;
}

void transpose(int *A, int *B, int rows, int columns)
{
    for(int rowId = 0; rowId < rows; rowId++ )
    {
        for(int columnId = 0; columnId < columns; columnId++)
        {
            // here we're basically saying that each column-major id for our matrix would be the 
            // main matrix's row major id, which in other words flips i, j to j, i correspondingly
            // the formula in itself is pretty smart when it comes to flattening a multidimensional matrix into 1d 
            // baically in row major order or rowId * column + columnId
            // we basically say jump to the row our element is at with rowId * columns, cus columns = number of elements in a row
            // when u multiply that with the column id u jump across that many columns to our column, once u jump there when u add
            // col id to the mix u basically jump to the id of our element within that row, thats basically it.
            B[columnId * rows + rowId] = A[rowId * columns + columnId]; 
        }
    }
}
int main(void)
{
    size_t size = N * N * sizeof(int);
    int *h_a = (int*)malloc(size);
    int *h_b = (int*)malloc(size);

    int *h_c = (int*)malloc(size);

    int *d_a, *d_c;

    for(int i = 0; i < N * N; i++)
    {
        h_a[i] = rand() % 100;
    }
    print_matrix(h_a, N, N, "Original Matrix A");
    transpose(h_a, h_b, N, N);

    print_matrix(h_b, N, N, "Transposed Matrix B");

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

    // initially my blockDim was kinda problematic with N, N, 0
    // Tomorrow we have to figure out how to break this thing, what if row, col weretn equal waht if it were greater what if it was lower eh and eh
    // figure out how stride loop would work for the kernel and so on, now im tired.
    // stick to hard core 256 and ceil for grid dim bingo

    // i think the gridDim and blockDim stuff is the most idiotic and problematic 

    // basically when u say launch a block with 16, 16 threads ur launching 256 threads in other words i want u to look at it like  amatrix
    // instead of looking at it 16 threads in x and y, in a 16 x 16 matrix although there are 16 threads in each direction
    // there are elements in that area which is 256 thats howthe threads are structured if u get what i mean 
    dim3 blockDim(16, 16, 1);

    // finally this is really important initially i was launching gridDim 1 dimensionally by ceil-ing for 256 threads directly across 1 dimension
    // why wont this work? essentially it wont cus look here threadIdx.y + blockDim.y * blockIdx.y;, when u do this itll launch blocks
    // across only x axis so blockDim across y axis will always be 0, so essentially when each block processes our matrix in 16 x 16 tiles
    // if our matrix goes beyond the 16 x 16 chunks it wont basically go beyond the first row/column depending on across what axis we're considering its still a bit hazy to me
    // might blackbox this
    dim3 gridDim(N+15/16, N+15/16, 1);  
    d_transpose<<<gridDim, blockDim>>>(d_a, d_c, N, N);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    print_matrix(h_c, N, N, "Transposed Matrix C");

    hipFree(d_a);
    hipFree(d_c);
    
    free(h_a);
    free(h_b);

    return 0;
}