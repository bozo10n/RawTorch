
#include <hip/hip_runtime.h>
#include <iostream>

# define N 10000

__global__ void add(int *a, int *b, int *c)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    while (id < N)
    {
        c[id] = a[id] + b[id];
        id += blockDim.x * gridDim.x;
    }
}

int main(void)
{
    int a[N], b[N], c[N];

    int *device_a, *device_b, *device_c;

    hipMalloc((void**)&device_a, N * sizeof(int));
    hipMalloc((void**)&device_b, N * sizeof(int));
    hipMalloc((void**)&device_c, N * sizeof(int));

    for(int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    hipMemcpy(device_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<128, 128>>>(device_a, device_b, device_c);


    hipMemcpy(c, device_c, N * sizeof(int), hipMemcpyDeviceToHost);


    for(int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}