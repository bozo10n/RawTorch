
#include <hip/hip_runtime.h>
# include <iostream>
# define count 1024


__global__ void gemmGPU(int *a, int *b, int *c, int M, int N, int K)
{
    int colId = threadIdx.x + blockIdx.x * blockDim.x;
    int rowId = threadIdx.y + blockIdx.y * blockDim.y;

    
    // this is wrong
    while(rowId < M && colId < N)
    {
        for(int k = 0; k < K; k++)
        {
            c[rowId * N + colId] += a[rowId * K + k] * b[k * N + colId];
        }
        rowId += blockDim.y * gridDim.y;

        colId += blockDim.x * gridDim.x;
    }
}
void printMatrix(int *a, int M, int N)
{
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++)
        {
            printf(" %d ", a[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void gemmCPU(int *a, int *b, int *c, int M, int N, int K)
{
    for(int i = 0; i < M; i++)
    {
        for(int j = 0; j < N; j++)
        {
            for(int k = 0; k < K; k++)
            {
                c[i * N + j] += a[i * K + k] * b[k * N + j];
            }
        }
    }
}

int main(void)
{
    int *a, *b, *c;

    size_t size = count * count * sizeof(int);

    a = (int*) malloc(size);
    b = (int*) malloc(size);
    c = (int*) malloc(size);

    for (int i = 0;  i < count * count; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    gemmCPU(a, b, c, count, count, count);

    int *d = (int*) malloc(size);

    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((count+15)/16, (count+15)/16);
    gemmGPU<<<gridDim, blockDim>>>(d_a, d_b, d_c, count, count, count);

    hipMemcpy(d, d_c, size, hipMemcpyDeviceToHost);

    if(memcmp(c, d, size) == 0)
    {
        printf("Winner winner chicken dinner!\n");
    }
    else
    {
        printf("Fuck you loser\n");
    }

    free(a);
    free(b);
    free(c);
    free(d);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;

}